#include "hip/hip_runtime.h"
#include "kernel.h"
//#include <stdio.h>
#include <hiprand.h>
#include <time.h>
#include <hipblas.h>

__global__ void KiELS2_kernel(
	optionData data1,
	optionData data2,
	double * d_s,
	double * stk, 
	double * payment, 
	double * date,
	double * d_normals,
	unsigned N_STEPS,
	unsigned N_SIMULS)
{
	int s_idx = threadIdx.x + blockIdx.x * blockDim.x; // thread index
	int n_idx = (s_idx) * N_STEPS; // for random number indexing	

	if (s_idx < N_SIMULS) {
		int n = 0;

		double s_curr1 = data1.S0, sigma1 = data1.sigma, r1 = data1.r, dt1 = data1.dt, sqrdt1 = data1.sqrdt, B1 = data1.B, dummy1 = data1.dummy;
		double s_curr2 = data2.S0, sigma2 = data2.sigma, r2 = data2.r, dt2 = data2.dt, sqrdt2 = data2.sqrdt, B2 = data2.B, dummy2 = data2.dummy;
		double ref_s1 = data1.S0_ref, ref_s2 = data2.S0_ref;

		double s_curr_cal1 = -1.0, s_curr_cal2 = -1.0;
		double s_curr_min = -1.0;
		double payoff = 0.0;
		unsigned int cnt1 = 0;
		unsigned int cnt2 = 0;
		double idx[length] = { 0 };

		double drift1 = (r1 - (sigma1*sigma1)*0.5)*dt1, sigsqdt1 = sigma1*sqrdt1;
		double drift2 = (r2 - (sigma2*sigma2)*0.5)*dt2, sigsqdt2 = sigma2*sqrdt2;

		bool tag = 0;
		bool kievent = 0;
		s_curr_cal1 = s_curr1 / ref_s1;
		s_curr_cal2 = s_curr2 / ref_s2;
		do {
			// Geometric Brownian motion
			s_curr_cal1 = s_curr_cal1 * exp(drift1 + sigsqdt1*d_normals[n_idx]);
			s_curr_cal2 = s_curr_cal2 * exp(drift2 + sigsqdt2*d_normals[N_STEPS*N_SIMULS + n_idx]);

			// worst performer
			s_curr_min = s_curr_cal1 < s_curr_cal2 ? s_curr_cal1 : s_curr_cal2;
			
			// cheeck knock-in event
			kievent = (s_curr_min < B1) ? 1 : kievent;

			// save underlying price at observation dates
			if ((n+1) == date[cnt1]) {
				idx[cnt1] = s_curr_min;
				cnt1++;
			}

			n_idx++;  // random number index
			n++;  // time stepping
		} while (n < N_STEPS);

		// check observation dates (early redemption)
		for (int i = 0; i < length; i++) {
			if (idx[i] >= stk[i]) {
				payoff = payment[i];
				tag = 1;
				cnt2 = i;
				break;
			}
		}
		if (tag == 0) {
			// payoff using ternary operator
			payoff = 10000 * s_curr_min;
			payoff = (kievent == 0) ? ((s_curr_min >= B1) ? 10000 * (1 + dummy1) : payoff) : payoff;
			cnt2 = length - 1;
		}

		payoff = payoff*exp(-r1 * date[cnt2] / 360.0);

		__syncthreads();
		d_s[s_idx] = payoff;
	}
}

void ELS2(
	optionData option1,
	optionData option2,
	double * d_s,
	double * stk,
	double * payment,
	double * date,
	double * d_normals,
	unsigned N_STEPS,
	unsigned N_SIMULS) {
	const unsigned BLOCK_SIZE = 1024; // # of threads in a block (1-dimension threads & block)
	const unsigned GRID_SIZE = CEIL(N_SIMULS, BLOCK_SIZE); // # of block in a grid
	KiELS2_kernel << <GRID_SIZE, BLOCK_SIZE >> >(
		option1, option2, d_s, stk, payment, date, d_normals, N_STEPS, N_SIMULS);
}
void dev_fillRand(double *A, size_t rows_A, size_t cols_A) 
{
	// random number generation host API
	hiprandGenerator_t rnd;
	hiprandCreateGenerator(&rnd, HIPRAND_RNG_PSEUDO_MTGP32);
	hiprandSetPseudoRandomGeneratorSeed(rnd, (unsigned long long)time(NULL));
	hiprandGenerateNormalDouble(rnd, A, rows_A*cols_A, 0.0, 1.0);
}

void dev_mmul(const double *A, const double *B, double *C, const int m, const int k, const int n) {
	// GPU matrix multiplication
    int lda = m,ldb = k,ldc = m;
    const double alf = 1;
    const double bet = 0;
    const double *alpha = &alf;
    const double *beta = &bet;

    // Create a handle for CUBLAS
    hipblasHandle_t handle;
    hipblasCreate(&handle);

    // Do the actual multiplication
    hipblasDgemm(handle, HIPBLAS_OP_N, HIPBLAS_OP_N, m, n, k, alpha, A, lda, B, ldb, beta, C, ldc);

    // Destroy the handle
    hipblasDestroy(handle);
}