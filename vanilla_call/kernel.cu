#include "hip/hip_runtime.h"
#include "kernel.h"

__global__ void Kernel_Vanilla_Call_single(
	optionData data,
	double * d_s,
	double * d_normals,
	unsigned N_STEPS,
	unsigned N_PATHS)
{
	int s_idx = threadIdx.x + blockIdx.x * blockDim.x; // i
	int n_idx = (s_idx)* N_STEPS;

	if (s_idx < N_PATHS) {
		int n = 0;

		double s_curr = data.S0;
		double T = data.T;
		double sig = data.sig;
		double r = data.r;
		double dt = data.dt;
		double sqrdt = data.sqrdt;
		double K = data.K;

		double payoff = 0.0;

		do {
			s_curr = s_curr * exp((r - (sig*sig)*0.5)*dt + sig*sqrdt*d_normals[n_idx]);
			
			n_idx++;
			n++;
		} while (n < N_STEPS);

		payoff = (s_curr > K)  ? (s_curr - K) : 0;

		__syncthreads();

		d_s[s_idx] = payoff;
	}
}

void Vanilla_Call_single(
	optionData option,
	double * d_s,
	double * d_normals,
	unsigned N_STEPS,
	unsigned N_PATHS) {
	const unsigned BLOCK_SIZE = 1024;
	const unsigned GRID_SIZE = CEIL(N_PATHS, BLOCK_SIZE);
	Kernel_Vanilla_Call_single << <GRID_SIZE, BLOCK_SIZE >> >
		(option, d_s, d_normals, N_STEPS, N_PATHS);
}